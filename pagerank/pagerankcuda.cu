#include "hip/hip_runtime.h"
/* Assignment 1 source code */

// #include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "libs/cudmatrix.h"

//constants
#define K 1000 // number of matvec iterations
#define MAX_BLOCK_SIZE 256

const float Q = .15;
//////////////////////////////////////////////////////////////////////////////////////////////////////////

/*Non parallel */
void minmaxPageRank(Vector *vec);
void dampen(DMatrix *H); // transform H matrix into G (dampened) matrix
/* parallel */
__global__ void d_normalize(float *d_v, int rowSize, int colSize, float *sum);
__global__ void d_mult(float *vec, float *mat, float *out, const int colSize, const int rowSize);
void vecNormalize(Vector *vec);            // normalize values of surfer values
Vector *matVec(DMatrix *mat, Vector *vec); // multiply compatible matrix and vector

int main(int argc, char *argv[])
{
    //reading number of pages from terminal
    uint numpg = (argc > 1) ? atoi(argv[1]) : 16;

    printf("-------Dense Matrix Test-----------------------\n\n");
    // create the H matrix
    DMatrix *H = initDMatrix(numpg);

    // create and initialize at the pagerank Vector
    Vector *pgrkV = initVector(numpg);

    clock_t startTime, endTime;

    // display the H matrix
    // printDMatrix(H);

    //prints pagerank vector before matvec
    // printf("pagerank vector before web surfing\n");
    // printDMatrix(pgrkV);

    startTime = clock();

    dampen(H);

    // apply matvec with dampening on for 1000 iterations
    for (uint iter = 0; iter < K; ++iter)
    {
        pgrkV = matVec(H, pgrkV); // parallelized matVecDampn
        // printf("pagerank after iter %d\n", iter);
        // printDMatrix(pgrkV);
    }

    if (numpg <= 16)
    { // print the page rank vector is small
        printf("pagerank vector after web surfing\n");
        printDMatrix(pgrkV);
    }

    // display lowest and highest page ranks
    minmaxPageRank(pgrkV);

    endTime = clock();
    printf("\nruntime = %.16e\n", ((float)(endTime - startTime)) / CLOCKS_PER_SEC);

    // garbage management
    destroyDMatrix(H);
    destroyDMatrix(pgrkV);

    return 0;
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////
// definition of dense matrix object
void minmaxPageRank(Vector *vec)
{
    // return the max and min values in the vector, as well as their indices

    float minval = vec->data[0 * vec->colSize + 0],
           maxval = vec->data[0 * vec->colSize + 0];

    uint minidx = 0, maxidx = 0;

    for (uint r = 0; r < vec->rowSize; ++r)
    {
        if (vec->data[r * vec->colSize + 0] >= maxval)
        {
            maxval = vec->data[r * vec->colSize + 0];
            maxidx = r;
        }

        if (vec->data[r * vec->colSize + 0] <= minval)
        {
            minval = vec->data[r * vec->colSize + 0];
            minidx = r;
        }
    }

    printf("X[min = %d] = %.6lf | X[max = %d] = %.6lf\n",
           minidx, minval, maxidx, maxval);
}

void dampen(DMatrix *m)
{
    // multiply compatible matrix and vector

    float numpg = m->colSize, tmp;

    for (uint r = 0; r < m->rowSize; ++r)
    {
        for (uint c = 0; c < m->colSize; ++c)
        {
            tmp = m->data[r * m->colSize + c];
            m->data[r * m->colSize + c] = (1 - Q) * tmp + Q / numpg;
        }
    }
}

///////////////////////////////////////////////////////////////////////
__global__ void 
d_normalize(float *d_vec, int rowSize, int colSize, float *d_sum)
{
    // normalize vector using gpu
    int index = threadIdx.x;
    int stride = blockDim.x;

    if (threadIdx.x == 0) *d_sum = 0;
    __syncthreads();

    for (int r = index; r < rowSize; r += stride)
    {
        atomicAdd(d_sum, d_vec[r * colSize + 0]);  
    }

    __syncthreads();

    for (int r = index; r < rowSize; r += stride)
    {
        d_vec[r * colSize + 0] /= *d_sum;
    }
       
}

void vecNormalize(Vector *vec)
{
    // normalize the content of vector

    //kernel call
    float *d_vec, *d_sum;

    //allocate space on device
    hipMalloc((void**)&d_sum, sizeof(float));
    hipMalloc((void**)&d_vec, sizeof(float) * vec->colSize * vec->rowSize);

    hipMemcpy(d_vec, vec->data,
        sizeof(float) * vec->colSize * vec->rowSize,
        hipMemcpyHostToDevice);
    
    int blocksPerGrid = 1;
    int threadsPerBlock = MAX_BLOCK_SIZE;

    d_normalize<<<blocksPerGrid, threadsPerBlock>>>(d_vec, vec->rowSize, vec->colSize, d_sum);
    // printf("d_normalize error code: %s\n",hipGetErrorString(hipGetLastError()));

    hipMemcpy(vec->data, d_vec, 
        sizeof(float) * vec->colSize * vec->rowSize,
        hipMemcpyDeviceToHost);

    // deallocate space from device
    hipFree(d_sum);
    hipFree(d_vec);
}

__global__
void d_mult(float *vec, float *mat, float *out, const int colSize, const int rowSize)
{
    int r = threadIdx.x + blockIdx.x * blockDim.x;
   
    float sum = 0.0;
    int vcolSize = 1;

    if(r < rowSize)
    {
        sum = 0.0;
        for(int c = 0; c < colSize; c++)
        {
            sum += mat[r * colSize + c] * vec[c * vcolSize + 0];
        }
        out[r * vcolSize + 0] = sum;
    }
 
}


Vector *matVec(DMatrix *m, Vector *vec)
{
    // multiply compatible matrix and vector
    // create and initialize at the pagerank Vector
    Vector *res = initVector(vec->rowSize);
    
    int threadsPerBlock = MAX_BLOCK_SIZE;
    int blocksPerGrid = m->rowSize / MAX_BLOCK_SIZE + 1;
    // int blocksPerGrid = 1;
    float *d_m, *d_vec, *d_res;

    //allocate space on device
    hipMalloc((void**)&d_m, sizeof(float) * m->rowSize * m->colSize);
    hipMalloc((void**)&d_vec, sizeof(float) * vec->rowSize * vec->colSize);
    hipMalloc((void**)&d_res, sizeof(float) * res->rowSize * res->colSize);


    hipMemcpy(d_m, m->data,
        sizeof(float)  * m->rowSize * m->colSize,
        hipMemcpyHostToDevice);

    hipMemcpy(d_vec, vec->data,
        sizeof(float) * vec->rowSize * vec->colSize,
        hipMemcpyHostToDevice);

    hipMemcpy(d_res, res->data,
        sizeof(float) * res->rowSize * res->colSize,
        hipMemcpyHostToDevice);

    d_mult<<<blocksPerGrid, threadsPerBlock>>>(d_vec, d_m, d_res, m->colSize, m->rowSize);   
    // printf("d_mult error code: %s\n",hipGetErrorString(hipGetLastError()));

    hipMemcpy(res->data, d_res, 
        sizeof(float) * res->rowSize * res->colSize,
        hipMemcpyDeviceToHost);

    // deallocate space from device
  
    hipFree(d_res);
    hipFree(d_vec);
    hipFree(d_m);

    // printDMatrix(res);


    vecNormalize(res);
    destroyDMatrix(vec);

    return res;
}
